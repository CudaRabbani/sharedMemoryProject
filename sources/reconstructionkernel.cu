#include "hip/hip_runtime.h"
#include "reconstruction.h"

//device functions definations

__device__ void StSPk_Operation(float *d_Vector, float *d_x, int *pattern);
__device__ void additionScalar(float *d_Vector, float *d_first, float *d_second, float scalar);
__device__ void multiplyA(float *d_Vector, float *device_x, float *d_x, int *pattern, float *convResult, int dataH, int dataW, float *temp);
__device__ void dotProduct(float *cache, float *temp);
__device__ void convolve(float *data, float *temp, float *convResult, int dataH, int dataW);
__device__ void dotProductSecond(float *cache, float *temp);

#define MASK_W 7
#define MASK_H 7
#define TILE_W 16 //It has to be same size as block
#define TILE_H 16 //It has to be same size as block
#define MASK_R (MASK_W / 2)

#define w (TILE_W + MASK_W -1)
#define clamp(x) (min(max((x), 0.0), 1.0))
#define ThreadPerBlock TILE_H*TILE_W


__constant__ float MASK[MASK_W * MASK_H];

__device__ __constant__ int Pix[TILE_H * TILE_W];

__global__ void reconstructionKernel(float *data, float *result, int *pattern, int dataH, int dataW, float *device_x, float *device_p)
{
    __shared__ float temp[w*w];
    __shared__ float convResult[ThreadPerBlock];
    __shared__ float d_Vector[ThreadPerBlock];
    __shared__ float d_current_x[ThreadPerBlock];
    __shared__ float d_current_r[ThreadPerBlock];
    __shared__ float d_current_p[ThreadPerBlock];
    __shared__ float d_next_x[ThreadPerBlock];
    __shared__ float d_next_r[ThreadPerBlock];
    __shared__ float d_next_p[ThreadPerBlock];
    __shared__ float cache_crnt_r[ThreadPerBlock]; //for dot product only
    __shared__ float cache_crnt_p[ThreadPerBlock]; //for dot product only
    __shared__ float cache_next_r[ThreadPerBlock]; //for dot product only
    __shared__ float cache[ThreadPerBlock];
    __shared__ int pixels[ThreadPerBlock];

    __shared__ float dot_Num;
    __shared__ float dot_Denom;
    __shared__ float dot_alpha;
    __shared__ float dot_beta;


    __shared__ float flag;
    __shared__ int counter;



    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;
    int localIndex = threadIdx.x + threadIdx.y * TILE_W;
    int index = tx + ty * dataW;
    int bid = blockIdx.x + blockIdx.y * gridDim.x;

    if((tx>=dataW) && (ty>=dataH))
        return;

    //Testing --------------------------------------------------------------------------------------------------
/*
    cache_crnt_p[localIndex] = index;
    cache_crnt_r[localIndex] = 1.0f;
    cache[localIndex] = cache_crnt_p[localIndex] * cache_crnt_r[localIndex];
    __syncthreads();
    result[blockIdx.x + blockIdx.y * gridDim.x] = dotProduct(cache);
*/
     //Testing --------------------------------------------------------------------------------------------------Ends
    if(localIndex == 0)
    {
        counter = 0;
        flag = 9.0f;
    }
    __syncthreads();


//    device_x[index] = 0.0f;
    d_current_x[localIndex] = device_x[index];//data[index];
    cache[localIndex] = data[index];
    pixels[localIndex] = pattern[index];
    __syncthreads();

//float *d_Vector, float *device_x, float *d_x, int* pattern, float *convResult, int dataH, int dataW, float *temp
    multiplyA(d_Vector, device_x, d_current_x, pixels, convResult, dataH, dataW, temp);
    __syncthreads();
    additionScalar(d_current_r, cache, d_Vector, -1); //cache = d_b; r = b - Ax
    __syncthreads();
    d_current_p[localIndex] = d_current_r[localIndex];
    __syncthreads();
    device_p[index] = d_current_p[localIndex];
    __syncthreads();

    // (fabs(flag - 0.00) > 1e-6) && (counter < 3) && (counter < 50)


        while ((counter < 30))
            {

                //Dot product goes here and the answer will be stored in dot_result_num
                cache_crnt_r[localIndex] = d_current_r[localIndex]*d_current_r[localIndex];
                __syncthreads();

                dotProduct(cache_crnt_r, &dot_Num);
                __syncthreads();


                multiplyA(d_Vector, device_p, d_current_p, pixels,convResult,dataH, dataW, temp);
                __syncthreads();

                cache_crnt_p[localIndex] = d_current_p[localIndex] * d_Vector[localIndex];
                __syncthreads();
        //        dot_result_denom = dotProduct(cache_crnt_p);


                dotProduct(cache_crnt_p, &dot_Denom);
                __syncthreads();

                if(localIndex == 0)
                {
                    dot_alpha = dot_Num / dot_Denom;
    //                printf("[%d] [alpha: %f] = %f/%f\n", bid, dot_alpha,dot_Num, dot_Denom);

                }
                __syncthreads();

                additionScalar(d_next_x, d_current_x, d_current_p, dot_alpha);
                __syncthreads();

                additionScalar(d_next_r, d_current_r,d_Vector, (-1)* dot_alpha);
                __syncthreads();


                __syncthreads();

                cache_next_r[localIndex] = d_next_r[localIndex] * d_next_r[localIndex];
                __syncthreads();
                dotProduct(cache_next_r, &dot_Denom); //beta = next_r/current_r
                __syncthreads();
                if(localIndex == 0)
                {
                    flag = sqrtf(dot_Denom);

                }
     //           flag = sqrtf(dot_Denom);
                __syncthreads();

                if(localIndex == 0)
                {
      //              printf("[%d]: %f\n", bid, dot_Denom);
                    dot_beta = dot_Denom / dot_Num;
                }
                 __syncthreads();

                additionScalar(d_next_p, d_next_r,d_current_p, dot_beta);
                 __syncthreads();


                d_current_r[localIndex] = d_next_r[localIndex];
                d_current_p[localIndex] = d_next_p[localIndex];
                d_current_x[localIndex] = d_next_x[localIndex];
                __syncthreads();
                device_p[index] = d_current_p[localIndex];
                __syncthreads();
                device_x[index] = d_current_x[localIndex];
                __syncthreads();
                if(localIndex == 0)
                {
                    counter = counter + 1;
                }
//                __syncthreads();
//                __threadfence();
                __threadfence_block();

            }
//        __syncthreads();
        __threadfence();
        __threadfence_block();

        result[index] = d_next_x[localIndex];
//             result[index] = cache[localIndex];

}
//  multiplyA(d_Vector, device_x, d_current_x, convResult, dataH, dataW, temp);
__device__ void multiplyA(float *d_Vector, float *device_x, float *d_x, int *pattern, float *convResult, int dataH, int dataW, float *temp)
{
    convolve(device_x, temp, convResult, dataH, dataW); //result will be also written on shared memory convResult;
    __syncthreads();
    StSPk_Operation(d_Vector, d_x, pattern); //result will be also stored on temp shared memory
    __syncthreads();
    additionScalar(d_Vector,d_Vector,convResult,1); //result will be stored in result
    __syncthreads();
//    prfloatf("%f\n", result[localIndex]);

}

__device__ void StSPk_Operation(float *d_Vector, float *d_x, int *pattern)
{
    int localIndex = threadIdx.x + threadIdx.y * TILE_W;
    d_Vector[localIndex] = d_x[localIndex] * pattern[localIndex];
//    printf("%f\n", Pix[localIndex]);
}

__device__ void additionScalar(float *d_Vector, float *d_first, float *d_second, float scalar)
{
    int localIndex = threadIdx.x + threadIdx.y * TILE_W;
    d_Vector[localIndex] = d_first[localIndex] + scalar*d_second[localIndex];
//    __syncthreads();
}


__device__ void dotProduct(float *cache, float *temp)
{

    int localIndex = threadIdx.x + threadIdx.y * blockDim.x;

    int i = ThreadPerBlock/2;

    while( (i!= 0) )
    {
        if(localIndex < i)
        {
            cache[localIndex] += cache[localIndex + i];

        }
        __syncthreads();
        i/=2;
    }
    __syncthreads();

    if(localIndex == 0)
    {
        temp[0] = cache[0];

    }


    __syncthreads();
//    return temp;


}



__device__ void dotProductSecond(float *cache, float *temp)
{

    int localIndex = threadIdx.x + threadIdx.y * blockDim.x;

    __shared__ int i;

    if(localIndex == 0)
    {
        i = ThreadPerBlock/2;
    }
    __syncthreads();


    while( (i!= 0) )
    {
        if(localIndex < i)
        {
            cache[localIndex] += cache[localIndex + i];

        }
        __syncthreads();
        if(localIndex == 0)
        {
             i/=2;
        }

        __syncthreads();
    }
    __syncthreads();

    if(localIndex == 0)
    {
        temp[0] = cache[0];
//        printf("dot: %f ", temp[0]);
//        return cache[0];
    }


//    __syncthreads();
//    return temp;


}




void reconstructionFunction(dim3 grid, dim3 block, float *data, int *pattern, float *kernel, float *d_result, int maskH, int maskW, int dataH, int dataW, float *device_x, float *device_p)
{

    if(hipMemcpyToSymbol(HIP_SYMBOL(MASK), kernel, maskH * maskW * sizeof(float)) != hipSuccess)
    {
        printf("Copy to constant memory error\n");
    }
    printf("Launching threads\n");
    reconstructionKernel<<<grid,block>>>(data, d_result, pattern, dataH, dataW, device_x, device_p);


}
/*
void initializePixelShuffle(float *d_pixels)
{
    if(hipMemcpyToSymbol(HIP_SYMBOL(Pix), d_pixels, sizeof(int) * 128 * 128) != hipSuccess)
    {
        printf("Writing of Pixel shuffle to constant memory Failed\n");
    }
    else
    {
        printf("Pixels are copied to constant memory\n");
    }
}
*/



/*
This convolve function reads data from global memory. Result is written back to Shared Memory

*/
__device__ void convolve(float *data, float *temp, float *convResult, int dataH, int dataW)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int localIndex;

    int dest = ty * TILE_W + tx;
    int destY = dest /w;
    int destX = dest % w;
//    printf("DEST: %d destX: %d destY: %d\n", dest, destX, destY);
    int srcY = by * TILE_W + destY - MASK_H/2;
    int srcX = bx * TILE_H + destX - MASK_W/2;
    int src = (srcY * dataW + srcX);
    localIndex = destY * w + destX;
//    printf("SOURCE: %d srcX: %d srcY: %d\n", src, srcX, srcY);
    if (srcY >= 0 && srcY < dataH && srcX >= 0 && srcX < dataW)
        temp[localIndex] = data[src];
//        temp[destY][destX] = data[src];
    else
        temp[localIndex] = 0.0;
//        temp[destY][destX] = 0.0;
    __syncthreads();
    dest = ty * TILE_W + tx + TILE_H * TILE_W;
    destY = dest / w;
    destX = dest % w;
    srcY = by * TILE_W + destY - MASK_R;
    srcX = bx * TILE_H + destX - MASK_R;
    src = srcY * dataW + srcX;
    localIndex = destY * w + destX;
    //    printf("DEST: %d destX: %d destY: %d\n", dest, destX, destY);
    //     printf("SOURCE: %d srcX: %d srcY: %d\n", src, srcX, srcY);
    if (destY < w)
    {
        if(srcY >= 0 && srcY <dataH && srcX >=0 && srcX <  dataW)
            temp[localIndex] = data[src];
//            temp[destY][destX] = data[src];
        else
            temp[localIndex] = 0.0;
//            temp[destY][destX] = 0.0;
    }


    __syncthreads();

    float out = 0.0f;
    int y,x;

    for(y = 0; y<MASK_H; y++)
    {
        for(x = 0; x<MASK_W; x++)
        {
            localIndex = (ty+y) * w + (tx+x);
            out += temp[localIndex] * MASK[y * MASK_W + x];
//            out += temp[ty + y][tx + x] * MASK[y * MASK_W + x];
        }
    }

    y = by * TILE_W + ty;
    x = bx * TILE_H + tx;

     localIndex = threadIdx.x + threadIdx.y * blockDim.x;
//    float tempY =threadIdx.y + blockIdx.y * blockDim.y;

    if(y < dataH && x < dataW)
    {
//        result[y * dataW + x] = out;
        convResult[localIndex] = out; //writing convolution result in shared memory for that block;
    }
//    prfloatf("%d\n", localIndex);
    __syncthreads();

}



//Not using these functions

__device__ float blockDotProduct(float *data_a,float *data_b, int width)
{

    float temp = 0;

    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;
    int tid = tidX + tidY * width;
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;

    __shared__ float cache[ThreadPerBlock]; //For Vector Dot Product


    int cachedIndex = threadIdx.x + threadIdx.y * blockDim.x;

    cache[cachedIndex] = data_a[tid]*data_b[tid];
    __syncthreads();
    if(cachedIndex==(ThreadPerBlock-1))
    {
        for(int i=0; i<ThreadPerBlock; i++)
        {
            temp += cache[i];

        }
    }
    return temp;

}
__global__ void  dotProductFunc(float *data_a, float *data_b, float *result, int dataH, int dataW)
{
    __shared__ float cache[ThreadPerBlock];

    float temp = 0;

    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;
    int tid = tidX + tidY * dataW;
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;

    int localIndex = threadIdx.x + threadIdx.y * blockDim.x;
    cache[localIndex] = data_a[tid]*data_b[tid];
    __syncthreads();
//    result[blockId] = dotProduct(cache);
 /*
    float i = ThreadPerBlock/2;

    while( (i!= 0) )
    {
        if(localIndex < i)
        {
            cache[localIndex] += cache[localIndex + i];

        }
        __syncthreads();
        i/=2;
    }

    if(localIndex == 0)
    {
        result[blockId] = cache[0];
    }
*/

}
void VectorDotProduct(dim3 gridSize, dim3 blockSize, float *data_a, float *data_b, float *d_result, int length, int width)
{
//    blockDotProduct<<<gridSize,blockSize>>>(data_a, data_b, d_result, result, length, width);
    dotProductFunc<<<gridSize,blockSize>>>(data_a, data_b, d_result, length, width);
//    Vector_Dot_Product<<<blockPerGrid,ThreadPerBlock>>>(data_a, data_b, d_result);
}

